// This file contains the implementation of the library defined by
// benchmark_gpu_utilities.h.
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "benchmark_gpu_utilities.h"

int InternalCUDAErrorCheck(hipError_t result, const char *fn,
  const char *file, int line) {
  if (result == hipSuccess) return 1;
  printf("CUDA error %d in %s, line %d (%s)\n", (int) result, file, line, fn);
  return 0;
}

hipError_t CreateCUDAStreamWithPriority(int stream_priority,
    hipStream_t *stream) {
  hipError_t result;
  int lowest_priority, highest_priority;
  result = hipDeviceGetStreamPriorityRange(&lowest_priority,
    &highest_priority);
  if (result != hipSuccess) return result;
  // Low priorities are higher numbers than high priorities.
  if ((stream_priority > lowest_priority) || (stream_priority <
    highest_priority)) {
    return hipStreamCreate(stream);
  }
  return hipStreamCreateWithPriority(stream, hipStreamNonBlocking,
    stream_priority);
}

